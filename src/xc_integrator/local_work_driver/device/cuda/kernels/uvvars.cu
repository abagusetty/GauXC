/**
 * GauXC Copyright (c) 2020-2024, The Regents of the University of California,
 * through Lawrence Berkeley National Laboratory (subject to receipt of
 * any required approvals from the U.S. Dept. of Energy). All rights reserved.
 *
 * See LICENSE.txt for details
 */
#include "device/common/uvvars.hpp"
#include "cuda_extensions.hpp"
#include <gauxc/util/div_ceil.hpp>

#include "uvvars_lda.hpp"
#include "uvvars_gga.hpp"
#include "uvvars_mgga.hpp"

namespace GauXC {

#define EVAL_UVARS_KERNEL(xc_approx) \
  hipStream_t stream = queue.queue_as<util::hip_stream>();  \
  switch ( ks_scheme ) { \
    case RKS: \
      eval_uvars_##xc_approx##_rks_kernel<<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case UKS: \
      eval_uvars_##xc_approx##_uks_kernel<<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case GKS: \
      eval_uvars_##xc_approx##_gks_kernel<<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    default: \
      GAUXC_GENERIC_EXCEPTION( "Unexpected KS scheme when attempting to evaluate U vars" ); \
  } 

#define EVAL_VVARS_KERNEL(xc_approx) \
  hipStream_t stream = queue.queue_as<util::hip_stream>();  \
  switch ( den_select ) { \
    case DEN_S: \
      eval_vvar_##xc_approx##_kern<DEN_S><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case DEN_Z: \
      eval_vvar_##xc_approx##_kern<DEN_Z><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case DEN_Y: \
      eval_vvar_##xc_approx##_kern<DEN_Y><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    case DEN_X: \
      eval_vvar_##xc_approx##_kern<DEN_X><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); \
      break; \
    default: \
      GAUXC_GENERIC_EXCEPTION( "Unexpected KS scheme when attempting to evaluate V vars" ); \
  } 

void eval_uvars_lda( size_t ntasks, int32_t npts_max, integrator_ks_scheme ks_scheme,
  XCDeviceTask* device_tasks, device_queue queue ) {
  dim3 threads( cuda::max_warps_per_thread_block * cuda::warp_size, 1, 1 );
  dim3 blocks( util::div_ceil( npts_max,  threads.x ), 1, ntasks ); 
  EVAL_UVARS_KERNEL(lda);
}

void eval_vvars_lda( size_t ntasks, int32_t nbf_max, int32_t npts_max, density_id den_select,
  XCDeviceTask* device_tasks, device_queue queue ) {
  dim3 threads( cuda::warp_size, cuda::max_warps_per_thread_block, 1 );
  dim3 blocks( util::div_ceil( nbf_max,  threads.x ),
               util::div_ceil( npts_max, threads.y ),
               ntasks );
  EVAL_VVARS_KERNEL(lda);

}






void eval_uvars_gga( size_t ntasks, int32_t npts_max, integrator_ks_scheme ks_scheme,
  XCDeviceTask* device_tasks, device_queue queue ) {
  dim3 threads( GGA_KERNEL_SM_WARPS * cuda::warp_size, 1, 1 );
  dim3 blocks( util::div_ceil( npts_max,  threads.x ), 1, ntasks ); 
  EVAL_UVARS_KERNEL(gga);
}

void eval_vvars_gga( size_t ntasks, int32_t nbf_max, int32_t npts_max, density_id den_select,
  XCDeviceTask* device_tasks, device_queue queue ) {
  dim3 threads( cuda::warp_size, cuda::max_warps_per_thread_block, 1 );
  dim3 blocks( util::div_ceil( nbf_max,  threads.x ),
               util::div_ceil( npts_max, threads.y ),
               ntasks );
  EVAL_VVARS_KERNEL(gga);
}


void eval_uvars_mgga( size_t ntasks, int32_t npts_max, integrator_ks_scheme ks_scheme,
  bool need_lapl, XCDeviceTask* device_tasks, device_queue queue ) {

  hipStream_t stream = queue.queue_as<util::hip_stream>(); 

  // Evaluate GAMMA
  eval_uvars_gga(ntasks, npts_max, ks_scheme, device_tasks, queue);

  if(ks_scheme == RKS) {
    return; // Nothing left to do
  } else if(ks_scheme == UKS) {
    dim3 threads( cuda::max_warps_per_thread_block * cuda::warp_size, 1, 1 );
    dim3 blocks( util::div_ceil( npts_max,  threads.x ), 1, ntasks ); 
    if(need_lapl) {
      eval_uvars_mgga_uks_kernel<true><<<blocks, threads, 0, stream>>>(ntasks, device_tasks);
    } else {
      eval_uvars_mgga_uks_kernel<false><<<blocks, threads, 0, stream>>>(ntasks, device_tasks);
    }
  } else {
    GAUXC_GENERIC_EXCEPTION("GKS + MGGA + DEVICE NYI");
  }

}

void eval_vvars_mgga( size_t ntasks, int32_t nbf_max, int32_t npts_max, density_id den_select,
  bool need_lapl, XCDeviceTask* device_tasks, device_queue queue ) {

  // First evaluate GGA variables
  eval_vvars_gga(ntasks, nbf_max, npts_max, den_select, device_tasks, queue);

  dim3 threads( cuda::warp_size, cuda::max_warps_per_thread_block, 1 );
  dim3 blocks( util::div_ceil( nbf_max,  threads.x ),
               util::div_ceil( npts_max, threads.y ),
               ntasks );

  hipStream_t stream = queue.queue_as<util::hip_stream>();
  switch ( den_select ) {
    case DEN_S:
      if(need_lapl)
        eval_vvar_mgga_kern<DEN_S,true><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); 
      else
        eval_vvar_mgga_kern<DEN_S,false><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); 
      break;
    case DEN_Z:
      if(need_lapl)
        eval_vvar_mgga_kern<DEN_Z,true><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); 
      else
        eval_vvar_mgga_kern<DEN_Z,false><<< blocks, threads, 0, stream >>>( ntasks, device_tasks ); 
      break;
    default: \
      GAUXC_GENERIC_EXCEPTION( "Unexpected KS scheme when attempting to evaluate V vars" ); \
  } 
}

}
